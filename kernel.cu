#include "hip/hip_runtime.h"
//
// Created by jzj2 on 2021/8/25.
//
#include "kernel.cuh"
#include <cstdio>
#include <iostream>


static void HandleError( hipError_t err, const char *file, int line )
{
    // CUDA error handeling from the "CUDA by example" book
    if (err != hipSuccess)
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

namespace kernel{

__global__ void cu_dot(Eigen::Vector3d *v1, Eigen::Vector3d *v2, double *out, size_t N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N)
    {
        out[idx] = v1[idx].dot(v2[idx]);
    }
    return;
}

// The wrapper for the calling of the actual kernel
double dot(const std::vector<Eigen::Vector3d> & v1, const std::vector<Eigen::Vector3d> & v2)
{
    // print some device info
    int dev = 0;
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, dev);
    std::cout << "using GPU device " << dev << ": " << devProp.name << std::endl;
    std::cout << "SM count " << devProp.multiProcessorCount << std::endl;
    std::cout << "sharedMemPerBlock " << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "maxThreadsPerBlock " << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "maxThreadsPerMultiProcessor " << devProp.maxThreadsPerMultiProcessor << std::endl;

    int n = v1.size();
    double *ret = new double[n];

    // Allocate device arrays
    Eigen::Vector3d *dev_v1, *dev_v2;
    HANDLE_ERROR(hipMalloc((void **)&dev_v1, sizeof(Eigen::Vector3d)*n));
    HANDLE_ERROR(hipMalloc((void **)&dev_v2, sizeof(Eigen::Vector3d)*n));
    double* dev_ret;
    HANDLE_ERROR(hipMalloc((void **)&dev_ret, sizeof(double)*n));

    // Copy to device
    HANDLE_ERROR(hipMemcpy(dev_v1, v1.data(), sizeof(Eigen::Vector3d)*n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_v2, v2.data(), sizeof(Eigen::Vector3d)*n, hipMemcpyHostToDevice));

    // Dot product
    cu_dot<<<(n+1023)/1024, 1024>>>(dev_v1, dev_v2, dev_ret, n);

    // Copy to host
    HANDLE_ERROR(hipMemcpy(ret, dev_ret, sizeof(double)*n, hipMemcpyDeviceToHost));

    // Reduction of the array
    for (int i=1; i<n; ++i)
    {
        ret[0] += ret[i];
    }

    // Return
    return ret[0];
}

}





